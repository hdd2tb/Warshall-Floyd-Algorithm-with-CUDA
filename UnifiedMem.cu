//calculate with Unified Memory

#include "include/gpgpu.h"
#include "include/Unified.h"
#include "include/Elapsed.h"

int main(){
	int n, m;
	Elapsed timer;
	cin >> n >> m;
	const int N = n * n;
	dim3 grid(n/128, n, 1), block(128, 1, 1);
	Unified<int> uni(N);

	executeDummy();

	uni.setData();

	timer.start();

	for (int k = 0; k < n; ++k)
		WarshallFloyd <<< grid, block >>> (n, k, uni.getData());

	hipDeviceSynchronize();

	timer.stop();

	timer.showElapsed();
	
	return 0;
}