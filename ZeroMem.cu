/*
calculate with Zero-Copy Memory
Zero-Copy Memoryを用いた実装
*/

#include "include/gpgpu.h"
#include "include/ZeroCopy.h"
#include "include/Elapsed.h"

int main(){
	int n, m;
	Elapsed timer;
	cin >> n >> m;
	const int N = n * n;
	dim3 grid(n/128, n, 1), block(128, 1, 1);
	ZeroCopy<int> zr(N);

	executeDummy();

	zr.setData();

	timer.start();

	for (int k = 0; k < n; ++k){
		WarshallFloyd <<< grid, block >>> (n, k, zr.getData());
		hipDeviceSynchronize();	
	}

	timer.stop();

	timer.showElapsed();

	return 0;
}