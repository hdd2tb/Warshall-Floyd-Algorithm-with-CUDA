//calculate with hipMalloc() and hipMemcpy()

#include "include/gpgpu.h"
#include "include/Adjacency.h"
#include "include/cuMall.h"
#include "include/Elapsed.h"

int main(){
	int n, m;
	Elapsed timer;
	cin >> n >> m;
	const int N = n * n;
	dim3 grid(n/128, n, 1), block(128, 1, 1);
	Adjacency<int> adj(N);
	cuMall<int> cu(N);

	executeDummy();

	adj.setData();
	cu.HostToDevice(adj.getData());

	timer.start();

	for (int k = 0; k < n; ++k)
		WarshallFloyd <<< grid, block >>> (n, k, cu.getData());

	hipDeviceSynchronize();

	cu.DeviceToHost(adj.getData());

	timer.stop();

	timer.showElapsed();

	return 0;	
}